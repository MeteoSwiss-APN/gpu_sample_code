#include <stdio.h>

int main() {
    int nDevices;
    
    hipGetDeviceCount(&nDevices);
    if (!nDevices)
    {
        printf("No CUDA capable devices found. Check your setup.");
        return 1;
    }

    printf("Detected %d CUDA devices\n", nDevices);
    for (int i = 0; i < nDevices; i++) {
        // Source: https://devblogs.nvidia.com/parallelforall/how-query-device-properties-and-handle-errors-cuda-cc/
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Compute Capability: sm_%d%d\n\n", prop.major, prop.minor);
    }
    return 0;
}
