#include  <stdio.h>
#include "hip/hip_runtime.h"

extern "C" {

	void saturation_adjustment_cuda(int ntot,
                                    double *t, double *qc, double *qv,
                                    double cs1, double cs2, double cs3, double cs4, double t0);

}

//--------------------------------------
// saturation adjustment kernel
__global__  void saturation_adjustment_kernel(int ntot,
                                              double *t, double *qc, double *qv,
                                              double cs1, double cs2, double cs3, double cs4, double t0)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( tid < ntot )
    {
        qv[tid] = qv[tid] + cs1 * exp( cs2 * ( t[tid] - t0 ) / ( t[tid] - cs3 ) );
        qc[tid] = cs4 * qv[tid];
    }
}

//--------------------------------------
// CUDA routine calling the saturation adjustment kernel
 void saturation_adjustment_cuda(int ntot,
                                 double *t, double *qc, double *qv,
                                 double cs1, double cs2, double cs3, double cs4, double t0)
{
    // set CUDA grid dimensions
    const int THREADS_PER_BLOCK = 128; // number of gpu threads per block
    const int NUMBER_OF_BLOCKS = ceil(ntot / THREADS_PER_BLOCK); 

    // calling CUDA kernel
    saturation_adjustment_kernel<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(ntot, t, qc, qv,
                                                                          cs1, cs2, cs3, cs4, t0);
}

